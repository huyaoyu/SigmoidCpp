#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

// ========== Device functions. ==========

template <typename scalar_t> 
__device__ __forceinline__ scalar_t d_sigmoid(scalar_t x)
{
    return 1.0 / ( 1.0 + exp(-x) );
}

// ========== Kernel functions. ==========

template <typename scalar_t>
__global__ void k_sigmoid_cpp_forward( 
    const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> input,
    torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> output )
{
    const int idxX    = blockIdx.x * blockDim.x + threadIdx.x;
    const int idxY    = blockIdx.y * blockDim.y + threadIdx.y;
    const int idxZ    = blockIdx.z * blockDim.z + threadIdx.z;
    const int strideX = gridDim.x * blockDim.x;
    const int strideY = gridDim.y * blockDim.y;
    const int strideZ = gridDim.z * blockDim.z;

    const int b = input.size(0);
    const int h = input.size(1);
    const int w = input.size(2);

    for ( int z = idxZ; z < b; z += strideZ )
    {
        for ( int y = idxY; y < h; y += strideY )
        {
            for ( int x = idxX; x < w; x += strideX )
            {
                output[z][y][x] = d_sigmoid( input[z][y][x] );
            }
        }
    }
}

template <typename scalar_t> 
__global__ void k_sigmoid_cpp_backward(
    const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> grad,
    const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> s,
    torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> output )
{
    const int idxX    = blockIdx.x * blockDim.x + threadIdx.x;
    const int idxY    = blockIdx.y * blockDim.y + threadIdx.y;
    const int idxZ    = blockIdx.z * blockDim.z + threadIdx.z;
    const int strideX = gridDim.x * blockDim.x;
    const int strideY = gridDim.y * blockDim.y;
    const int strideZ = gridDim.z * blockDim.z;

    const int b = s.size(0);
    const int h = s.size(1);
    const int w = s.size(2);

    for (int z = idxZ; z < b; z += strideZ )
    {
        for ( int y = idxY; y < h; y += strideY )
        {
            for ( int x = idxX; x < w; x += strideX )
            {
                output[z][y][x] = 
                    grad[z][y][x] * 
                    ( 1.0 - s[z][y][x] ) * s[z][y][x];
            }
        }
    }
}

// ========== Interface functions. ==========

std::vector<torch::Tensor> sigmoid_cpp_forward_cuda( torch::Tensor input )
{
    // Get the batch size.
    auto b = input.size(0);

    // The 2D tensor dimensions.
    auto h = input.size(1);
    auto w = input.size(2);

    // Prepare output.
    auto output = torch::zeros_like(input);

    const int threadsX = 2;
    const int threadsY = 2;

    // Kernal launch dimensions.
    const dim3 blocks( ( w + threadsX - 1 ) / threadsX, ( h + threadsY - 1 ) / threadsY, b );
    const dim3 thrds( threadsX, threadsY, 1 );

    // Kernal launch.
    AT_DISPATCH_FLOATING_TYPES( input.type(), "sigmoid_cpp_forwrd_cuda", ([&] {
        k_sigmoid_cpp_forward<scalar_t><<<blocks, thrds>>>( 
            input.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
            output.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>() );
    }) );

    return { output };
}

std::vector<torch::Tensor> sigmoid_cpp_backward_cuda( torch::Tensor grad, torch::Tensor s )
{
    // Get the batch size.
    auto b = s.size(0);

    // Get the 2D tensor dimesnions.
    auto h = s.size(1);
    auto w = s.size(2);

    // The result.
    auto output = torch::zeros_like(s);

    const int threadsX = 2;
    const int threadsY = 2;

    // Kernal launch dimensions.
    const dim3 blocks( ( w + threadsX - 1 ) / threadsX, ( h + threadsY - 1 ) / threadsY, b );
    const dim3 thrds( threadsX, threadsY, 1 );

    // Kernal launch.
    AT_DISPATCH_FLOATING_TYPES( s.type(), "sigmoid_cpp_backward_cuda", ( [&] {
        k_sigmoid_cpp_backward<scalar_t><<<blocks, thrds>>>( 
            grad.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
            s.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
            output.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>() );
    } ) );

    return { output };
}
